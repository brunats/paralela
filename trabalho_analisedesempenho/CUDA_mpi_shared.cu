
#include <hip/hip_runtime.h>
#include <stdio.h>

//compilar: nvcc matrizMultiplicacaoCompartilhada.cu -o matrizMultiplicacaoComp
//for i in `seq 1 10`; do ./matrizMultiplicacaoComp; done

//#define N 64
#define B 16
#define TILE_WIDTH 16

extern double *mA, *mB, *mC;
extern int N, num;

__global__ void matrixMulKernel(double *m, double *n, double *p, int width)
{
    __shared__ double sm[TILE_WIDTH][TILE_WIDTH];
    __shared__ double sn[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    double pvalue = 0;

    for (int i = 0; i < width/TILE_WIDTH; i++) {
        sm[threadIdx.y][threadIdx.x] = m[row * width + (i * TILE_WIDTH + threadIdx.x)];
        sn[threadIdx.y][threadIdx.x] = n[col + (i * TILE_WIDTH + threadIdx.y) * width];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            pvalue += sm[threadIdx.y][k] * sn[k][threadIdx.x];
        }
        __syncthreads();

    }
    p[row * width + col] = pvalue;
}

extern "C" void multiplica() {
	//double *a, *b, *c;
	double *d_a, *d_b, *d_c;
	int size = N;
	dim3 dimen (B, B);
	//cudaEvent_t start, stop;
	
	//cudaEventCreate(&start);
	//cudaEventCreate(&stop);

	hipMalloc( (void **) &d_a, (size/num)*size*sizeof(double) );
	hipMalloc( (void **) &d_b, size*size*sizeof(double) );
	hipMalloc( (void **) &d_c, (size/num)*size*sizeof(double) );

	//a = (double *)malloc( size*size*sizeof(double) );
	//b = (double *)malloc( size*size*sizeof(double) );
	//c = (double *)malloc( size*size*sizeof(double) );

	for( int i = 0; i < (N/num)*N; i++ ) {
		//a[i] = b[i] = i;
		mC[i] = 0;
	}
    
    //cudaEventRecord(start);
    
	hipMemcpy( d_a, mA, (size/num)*size*sizeof(double), hipMemcpyHostToDevice );
	hipMemcpy( d_b, mB, size*size*sizeof(double), hipMemcpyHostToDevice );

	dim3 grade ((N + B-1)/B, ((N/num) + B-1)/B);
	matrixMulKernel<<<grade, dimen>>>( d_a, d_b, d_c, N );

	hipMemcpy( mC, d_c, (size/num)*size*sizeof(double), hipMemcpyDeviceToHost );
	
	
	//cudaEventRecord(stop);
	//cudaEventSynchronize(stop);
    //double milliseconds = 0;
    //cudaEventElapsedTime(&milliseconds, start, stop);
    
    //printf("%f\n", milliseconds/1000.0);
    
    
	//printf( "c[0] = %lf\n", c[0] );
	//printf( "c[%d] = %lf\n",N*N, c[N*N-1] );
	
	/*
	int i;
	for(i=0; i<N*N; i++){
		printf( "c[%d] = %lf\n",i, c[i] );
	}
	*/
	

	//free(a);
	//free(b);
	//free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	//return 0;
} /* end main */
