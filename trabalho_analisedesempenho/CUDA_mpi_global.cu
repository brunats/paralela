
#include <hip/hip_runtime.h>
#include <stdio.h>

//compilar: nvcc matrizMultiplicacao.cu -o matrizMultiplicacao
//for i in `seq 1 10`; do ./matrizMultiplicacao; done

//#define N 64
#define B 16

extern double *mA, *mB, *mC;
extern int N, num;

__global__ void matrix_multi(double *a, double *b, double *c, int N) {
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int i;
	double soma = 0.0;
	if(x < N && y < N){
		for(i=0; i<N; i++){
			soma += a[x * N + i ] * b[y + N * i];
		}
		c[y + x * N] = soma;
	}
}

extern "C" void multiplica() {
	//double *c;
	double *d_a, *d_b, *d_c;
	int size = N;
	dim3 dimen (B, B);
	//int num = 2;
	//cudaEvent_t start, stop;
	
	//cudaEventCreate(&start);
	//cudaEventCreate(&stop);

	hipMalloc( (void **) &d_a, (size/num)*size*sizeof(double) );
	hipMalloc( (void **) &d_b, size*size*sizeof(double) );
	hipMalloc( (void **) &d_c, (size/num)*size*sizeof(double) );

	//a = (double *)malloc( (size/num)*size*sizeof(double) );
	//b = (double *)malloc( size*size*sizeof(double) );
	//c = (double *)malloc( (size/num)*size*sizeof(double) );

	/*
	for( int i = 0; i < N*N; i++ ) {
		//a[i] = b[i] = i;
		b[i] = i;
	}*/
	for(int i = 0; i < (N/num)*N; i++){
	    //a[i] = i;
	    mC[i] = 0;
	}
    
    //cudaEventRecord(start);
    
	hipMemcpy( d_a, mA, (size/num)*size*sizeof(double), hipMemcpyHostToDevice );
	hipMemcpy( d_b, mB, size*size*sizeof(double), hipMemcpyHostToDevice );

	//dim3 grade (((N/num) + B-1)/B, (N + B-1)/B);
	dim3 grade ((N + B-1)/B, ((N/num) + B-1)/B);
	matrix_multi<<<grade, dimen>>>( d_a, d_b, d_c , N );

	hipMemcpy( mC, d_c, (size/num)*size*sizeof(double), hipMemcpyDeviceToHost );
	
	/*
	cudaEventRecord(stop);
	cudaEventSynchronize(stop);
    double milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    
    printf("%f\n", milliseconds/1000.0);
    */
    
	//printf( "c[0] = %lf\n", c[0] );
	//printf( "c[%d] = %lf\n",N*N, c[N*N-1] );
	
	/*
	int i;
	for(i=0; i<(N/num)*N; i++){
		printf( "c[%d] = %lf\n",i, c[i] );
	}
	*/
	

	//free(a);
	//free(b);
	//free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

}
