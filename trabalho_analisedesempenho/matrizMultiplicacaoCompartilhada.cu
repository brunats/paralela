
#include <hip/hip_runtime.h>
#include <stdio.h>

//compilar: nvcc matrizMultiplicacaoCompartilhada.cu -o matrizMultiplicacaoComp
//for i in `seq 1 10`; do ./matrizMultiplicacaoComp; done

#define N 64
#define B 32
#define TILE_WIDTH 32

__global__ void matrix_multi(float *a, float *b, float *c) {
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int i;
	float soma = 0.0;
	if(x < N && y < N){
		for(i=0; i<N; i++){
			soma += a[x * N + i ] * b[y + N * i];
		}
		c[x + y * N] = soma;
	}
}

__global__ void matrixMulKernel(float *m, float *n, float *p, int width)
{
    __shared__ float sm[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sn[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float pvalue = 0;

    for (int i = 0; i < width/TILE_WIDTH; i++) {
        sm[threadIdx.y][threadIdx.x] = m[row * width + (i * TILE_WIDTH + threadIdx.x)];
        sn[threadIdx.y][threadIdx.x] = n[col + (i * TILE_WIDTH + threadIdx.y) * width];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            pvalue += sm[threadIdx.y][k] * sn[k][threadIdx.x];
        }
        __syncthreads();

    }
    p[row * width + col] = pvalue;
}

int main() {
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int size = N;
	dim3 dimen (B, B);
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc( (void **) &d_a, size*size*sizeof(float) );
	hipMalloc( (void **) &d_b, size*size*sizeof(float) );
	hipMalloc( (void **) &d_c, size*size*sizeof(float) );

	a = (float *)malloc( size*size*sizeof(float) );
	b = (float *)malloc( size*size*sizeof(float) );
	c = (float *)malloc( size*size*sizeof(float) );

	for( int i = 0; i < N*N; i++ ) {
		a[i] = b[i] = i;
		c[i] = 0;
	}
    
    hipEventRecord(start);
    
	hipMemcpy( d_a, a, size*size*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size*size*sizeof(float), hipMemcpyHostToDevice );

	dim3 grade ((N + B-1)/B, (N + B-1)/B);
	matrixMulKernel<<<grade, dimen>>>( d_a, d_b, d_c, N );

	hipMemcpy( c, d_c, size*size*sizeof(float), hipMemcpyDeviceToHost );
	
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("%f\n", milliseconds/1000.0);
    
    
	//printf( "c[0] = %lf\n", c[0] );
	//printf( "c[%d] = %lf\n",N*N, c[N*N-1] );
	
	int i;
	for(i=0; i<N*N; i++){
		printf( "c[%d] = %lf\n",i, c[i] );
	}
	
	

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	return 0;
} /* end main */
