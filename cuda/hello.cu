
#include <hip/hip_runtime.h>
#include<stdio.h>

//compilar: nvcc hello.cu -o hello

__device__ const char *STR = "Hello world!";
const char STR_LENGHT = 12;

__global__ void hello(){
    printf("%c\n", STR[threadIdx.x % STR_LENGHT]);
}

int main(void){
    int num_threads = STR_LENGHT;
    int num_blocks = 2;
    dim3 dimBlock(16,16);
    dim3 dimGrid(32,32);
    //hello<<<dimGrid,dimBlock>>>();
    hello<<<1,12>>>();
    hipDeviceSynchronize();
    
    return 0;
}
