
#include <hip/hip_runtime.h>
#include <stdio.h>

//compilar: nvcc matrizMultiplicacao.cu -o matrizMultiplicacao

#define N 3
#define B 1

__global__ void matrix_multi(float *a, float *b, float *c) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int i;
	float soma = 0.0;
	if(x < N && y < N){
		for(i=0; i<N; i++){
			soma += a[x * i + y ] * b[x + y * i];
		}
		c[x + y * N] = soma;
	}
}

int main() {
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int size = N;
	dim3 dimen (B, B);

	hipMalloc( (void **) &d_a, size*size*sizeof(float) );
	hipMalloc( (void **) &d_b, size*size*sizeof(float) );
	hipMalloc( (void **) &d_c, size*size*sizeof(float) );

	a = (float *)malloc( size*size*sizeof(float) );
	b = (float *)malloc( size*size*sizeof(float) );
	c = (float *)malloc( size*size*sizeof(float) );

	for( int i = 0; i < N*N; i++ ) {
		a[i] = b[i] = 1;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, size*size*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size*size*sizeof(float), hipMemcpyHostToDevice );

	dim3 grade ((N + B-1)/B, (N + B-1)/B);
	matrix_multi<<<grade, dimen>>>( d_a, d_b, d_c );

	hipMemcpy( c, d_c, size*size*sizeof(float), hipMemcpyDeviceToHost );

	int i;
	for(i=0; i<N*N; i++){
		printf( "c[%d] = %lf\n",i, c[i] );
	}

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	return 0;
} /* end main */
