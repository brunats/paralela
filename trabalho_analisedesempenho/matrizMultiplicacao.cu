
#include <hip/hip_runtime.h>
#include <stdio.h>

//compilar: nvcc matrizMultiplicacao.cu -o matrizMultiplicacao
//for i in `seq 1 10`; do ./matrizMultiplicacao; done

#define N 64
#define B 32

__global__ void matrix_multi(float *a, float *b, float *c) {
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int i;
	float soma = 0.0;
	if(x < N && y < N){
		for(i=0; i<N; i++){
			soma += a[x * N + i ] * b[y + N * i];
		}
		c[y + x * N] = soma;
	}
}

int main() {
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int size = N;
	dim3 dimen (B, B);
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc( (void **) &d_a, size*size*sizeof(float) );
	hipMalloc( (void **) &d_b, size*size*sizeof(float) );
	hipMalloc( (void **) &d_c, size*size*sizeof(float) );

	a = (float *)malloc( size*size*sizeof(float) );
	b = (float *)malloc( size*size*sizeof(float) );
	c = (float *)malloc( size*size*sizeof(float) );

	for( int i = 0; i < N*N; i++ ) {
		a[i] = b[i] = i;
		c[i] = 0;
	}
    
    hipEventRecord(start);
    
	hipMemcpy( d_a, a, size*size*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size*size*sizeof(float), hipMemcpyHostToDevice );

	dim3 grade ((N + B-1)/B, (N + B-1)/B);
	matrix_multi<<<grade, dimen>>>( d_a, d_b, d_c );

	hipMemcpy( c, d_c, size*size*sizeof(float), hipMemcpyDeviceToHost );
	
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("%f\n", milliseconds/1000.0);
    
    
	//printf( "c[0] = %lf\n", c[0] );
	//printf( "c[%d] = %lf\n",N*N, c[N*N-1] );
	
	int i;
	for(i=0; i<N*N; i++){
		printf( "c[%d] = %lf\n",i, c[i] );
	}
	
	

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	return 0;
} /* end main */
