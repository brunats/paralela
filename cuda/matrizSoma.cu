
#include <hip/hip_runtime.h>
#include <stdio.h>

//compilar: nvcc matrizSoma.cu -o matrizSoma

#define N 512
#define B 32

__global__ void matrix_add(float *a, float *b, float *c) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x < N && y < N){
		c[x + y * N] = a[x + y * N] + b[x + y * N];
	}
}

int main() {
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int size = N;
	dim3 dimen (B, B);

	hipMalloc( (void **) &d_a, size*size*sizeof(float) );
	hipMalloc( (void **) &d_b, size*size*sizeof(float) );
	hipMalloc( (void **) &d_c, size*size*sizeof(float) );

	a = (float *)malloc( size*size*sizeof(float) );
	b = (float *)malloc( size*size*sizeof(float) );
	c = (float *)malloc( size*size*sizeof(float) );

	for( int i = 0; i < N*N; i++ ) {
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, size*size*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size*size*sizeof(float), hipMemcpyHostToDevice );


	dim3 grade ((N + B-1)/B, (N + B-1)/B);

	matrix_add<<<grade, dimen>>>( d_a, d_b, d_c );

	hipMemcpy( c, d_c, size*size*sizeof(float), hipMemcpyDeviceToHost );

	int i;
	for(i=0; i<N*N; i++){
		printf( "c[%d] = %lf\n",i, c[i] );
	}

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	return 0;
} /* end main */
